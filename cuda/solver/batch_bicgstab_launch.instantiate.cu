#include "hip/hip_runtime.h"
// SPDX-FileCopyrightText: 2017 - 2024 The Ginkgo authors
//
// SPDX-License-Identifier: BSD-3-Clause

#include "cuda/solver/batch_bicgstab_launch.cuh"

#include <ginkgo/core/base/exception_helpers.hpp>

#include "common/cuda_hip/solver/batch_bicgstab_kernels.hpp"
#include "core/matrix/batch_struct.hpp"
#include "core/solver/batch_bicgstab_kernels.hpp"
#include "core/solver/batch_dispatch.hpp"


namespace gko {
namespace kernels {
namespace cuda {
namespace batch_bicgstab {


template <typename StopType, typename PrecType, typename LogType,
          typename BatchMatrixType, typename ValueType>
int get_num_threads_per_block(std::shared_ptr<const DefaultExecutor> exec,
                              const int num_rows)
{
    int num_warps = std::max(num_rows / 4, 2);
    constexpr int warp_sz = static_cast<int>(config::warp_size);
    const int min_block_size = 2 * warp_sz;
    const int device_max_threads =
        ((std::max(num_rows, min_block_size)) / warp_sz) * warp_sz;
    hipFuncAttributes funcattr;
    hipFuncGetAttributes(
        &funcattr,
        reinterpret_cast<const void*>(batch_single_kernels::apply_kernel<StopType), 9, true, PrecType, LogType,
                                           BatchMatrixType, ValueType>);
    const int num_regs_used = funcattr.numRegs;
    int max_regs_blk = 0;
    hipDeviceGetAttribute(&max_regs_blk, hipDeviceAttributeMaxRegistersPerBlock,
                           exec->get_device_id());
    const int max_threads_regs =
        ((max_regs_blk / static_cast<int>(num_regs_used)) / warp_sz) * warp_sz;
    int max_threads = std::min(max_threads_regs, device_max_threads);
    max_threads = max_threads <= 1024 ? max_threads : 1024;
    return std::max(std::min(num_warps * warp_sz, max_threads), min_block_size);
}


template <typename StopType, typename PrecType, typename LogType,
          typename BatchMatrixType, typename ValueType>
int get_max_dynamic_shared_memory(std::shared_ptr<const DefaultExecutor> exec)
{
    int shmem_per_sm = 0;
    hipDeviceGetAttribute(&shmem_per_sm,
                           hipDeviceAttributeMaxSharedMemoryPerMultiprocessor,
                           exec->get_device_id());
    GKO_ASSERT_NO_CUDA_ERRORS(hipFuncSetAttribute(reinterpret_cast<const void*>(
        batch_single_kernels::apply_kernel<StopType), 9, true, PrecType, LogType,
                                           BatchMatrixType, ValueType>,
        hipFuncAttributePreferredSharedMemoryCarveout, 99 /*%*/));
    hipFuncAttributes funcattr;
    hipFuncGetAttributes(
        &funcattr,
        reinterpret_cast<const void*>(batch_single_kernels::apply_kernel<StopType), 9, true, PrecType, LogType,
                                           BatchMatrixType, ValueType>);
    return funcattr.maxDynamicSharedSizeBytes;
}


template <typename ValueType, int n_shared, bool prec_shared, typename StopType,
          typename PrecType, typename LogType, typename BatchMatrixType>
void launch_apply_kernel(
    std::shared_ptr<const DefaultExecutor> exec,
    const gko::kernels::batch_bicgstab::storage_config& sconf,
    const settings<remove_complex<ValueType>>& settings, LogType& logger,
    PrecType& prec, const BatchMatrixType& mat,
    const ValueType* const __restrict__ b_values,
    ValueType* const __restrict__ x_values,
    ValueType* const __restrict__ workspace_data, const int& block_size,
    const size_t& shared_size)
{
    batch_single_kernels::apply_kernel<StopType, n_shared, prec_shared>
        <<<mat.num_batch_items, block_size, shared_size, exec->get_stream()>>>(
            sconf, settings.max_iterations, as_cuda_type(settings.residual_tol),
            logger, prec, mat, b_values, x_values, workspace_data);
}


// begin
GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(
    GKO_DECLARE_BATCH_BICGSTAB_GET_NUM_THREADS_PER_BLOCK);
// split
GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(
    GKO_DECLARE_BATCH_BICGSTAB_GET_MAX_DYNAMIC_SHARED_MEMORY);
// split
GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(GKO_DECLARE_BATCH_BICGSTAB_LAUNCH_0_FALSE);
// split
GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(GKO_DECLARE_BATCH_BICGSTAB_LAUNCH_1_FALSE);
// split
GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(GKO_DECLARE_BATCH_BICGSTAB_LAUNCH_2_FALSE);
// split
GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(GKO_DECLARE_BATCH_BICGSTAB_LAUNCH_3_FALSE);
// split
GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(GKO_DECLARE_BATCH_BICGSTAB_LAUNCH_4_FALSE);
// split
GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(GKO_DECLARE_BATCH_BICGSTAB_LAUNCH_5_FALSE);
// split
GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(GKO_DECLARE_BATCH_BICGSTAB_LAUNCH_6_FALSE);
// split
GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(GKO_DECLARE_BATCH_BICGSTAB_LAUNCH_7_FALSE);
// split
GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(GKO_DECLARE_BATCH_BICGSTAB_LAUNCH_8_FALSE);
// split
GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(GKO_DECLARE_BATCH_BICGSTAB_LAUNCH_9_FALSE);
// split
GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(GKO_DECLARE_BATCH_BICGSTAB_LAUNCH_9_TRUE);
// end


}  // namespace batch_bicgstab
}  // namespace cuda
}  // namespace kernels
}  // namespace gko
